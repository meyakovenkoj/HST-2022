#include "hip/hip_runtime.h"
extern "C" {
#include <stdio.h>
#include "cuda_process.h"
}

#define CUDA_ERROR() \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err) { \
            printf("File: %s, Line: %d, error code: %d\n", __FILE__, __LINE__, __err); \
            return; \
        } \
    } while(0)

__global__ void cosine(const float *array1, const float *array2, int count, int length, float *result)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= count) {
        return;
    }
    const float *first_arr = array1 + tid * length;
    const float *second_arr = array2 + tid * length;
    float uv = 0;
    float uu = 0;
    float vv = 0;
    for (int i = 0; i < length; i++) {
        uv += first_arr[i] * second_arr[i];
        uu += first_arr[i] * first_arr[i];
        vv += second_arr[i] * second_arr[i];
    }
    result[tid] = uv / (sqrtf(uu) * sqrtf(vv));
}

void gpu_process(float *array1, float *array2, int count, int length, float *result, float **dev_array1, float **dev_array2, float **dev_result)
{
    hipMalloc((void**)dev_array1, count * length * sizeof(float));
    CUDA_ERROR();
    hipMalloc((void**)dev_array2, count * length * sizeof(float));
    CUDA_ERROR();
    hipMalloc((void**)dev_result, count * sizeof(float));
    CUDA_ERROR();

    hipMemcpyAsync(*dev_array1, array1, count * length * sizeof(float), hipMemcpyHostToDevice);
    CUDA_ERROR();
    hipMemcpyAsync(*dev_array2, array2, count * length * sizeof(float), hipMemcpyHostToDevice);
    CUDA_ERROR();

    int block_size = 1024;
    int grid_size = (count + block_size) / block_size;
    cosine<<<grid_size,block_size>>>(*dev_array1, *dev_array2, count, length, *dev_result);

    hipMemcpyAsync(result, *dev_result, sizeof(float) * count, hipMemcpyDeviceToHost);
    CUDA_ERROR();
}

void gpu_clean_and_sync(float *dev_array1, float *dev_array2, float *dev_result)
{
    hipDeviceSynchronize();
    hipFree(dev_array1);
    hipFree(dev_array2);
    hipFree(dev_result);
}
