#include "hip/hip_runtime.h"
extern "C" {
#include "cuda_process.h"
}

__global__ void cosine(const float *array1, const float *array2, int count, int length, float *result)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    const float *first_arr = array1 + tid * length;
    const float *second_arr = array2 + tid * length;
    float uv = 0;
    float uu = 0;
    float vv = 0;
    for (int i = 0; i < length; i++) {
        uv += first_arr[i] * second_arr[i];
        uu += first_arr[i] * first_arr[i];
        vv += second_arr[i] * second_arr[i];
    }
    result[blockIdx.x] = uv / (sqrtf(uu) * sqrtf(vv));
}

void gpu_process(float *array1, float *array2, int count, int length, float *result, float **dev_array1, float **dev_array2, float **dev_result)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    hipMalloc((void**)dev_array1, count * length * sizeof(float));
    hipMalloc((void**)dev_array2, count * length * sizeof(float));
    hipMalloc((void**)dev_result, count * sizeof(float));

    hipMemcpyAsync(*dev_array1, array1, count * length * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(*dev_array2, array2, count * length * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 1024;
    int grid_size = (count + block_size) / block_size;
    cosine<<<grid_size,block_size>>>(*dev_array1, *dev_array2, count, length, *dev_result);

    hipMemcpyAsync(result, *dev_result, sizeof(float) * count, hipMemcpyDeviceToHost);
}

void gpu_clean_and_sync(float *dev_array1, float *dev_array2, float *dev_result)
{
    hipDeviceSynchronize();
    hipFree(dev_array1);
    hipFree(dev_array2);
    hipFree(dev_result);
}